#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <conio.h>
#include <tchar.h>
#include <time.h>
#include <stdbool.h>

#define ELEM_U 0
#define ELEM_V 1
#define ELEM_DENS 2
#define ELEM_NUM 3
//#define ENGINE_STARTED 1.01
//#define FRAME_FINISHED 1.02
#define STOP_ENGINE 1.0
#define NEXT_FRAME 2.0
#define NO_MESSAGE 0.0
//#define FRAME_DISPLAYED 0.02
/*#define swap(fp1, fp2)    \
    {                     \
        float *tmp = fp1; \
        fp1 = fp2;        \
        fp1 = tmp;        \
    }*/
#define IDX(c, r, e) (((c) + (size + 2) * (r)) * ELEM_NUM + (e))

__device__ void set_bnd(int size, int b, float *matrix, int elem)
{
    int i;
    for (i = 1; i <= size; i++)
    {
        matrix[IDX(0, i, elem)] = b == 1 ? -matrix[IDX(1, i, elem)] : matrix[IDX(1, i, elem)];
        matrix[IDX(size + 1, i, elem)] = b == 1 ? -matrix[IDX(size, i, elem)] : matrix[IDX(size, i, elem)];
        matrix[IDX(i, 0, elem)] = b == 2 ? -matrix[IDX(i, 1, elem)] : matrix[IDX(i, 1, elem)];
        matrix[IDX(i, size + 1, elem)] = b == 2 ? -matrix[IDX(i, size, elem)] : matrix[IDX(i, size, elem)];
    }
    matrix[IDX(0, 0, elem)] = 0.5 * (matrix[IDX(1, 0, elem)] + matrix[IDX(0, 1, elem)]);
    matrix[IDX(0, size + 1, elem)] = 0.5 * (matrix[IDX(1, size + 1, elem)] + matrix[IDX(0, size, elem)]);
    matrix[IDX(size + 1, 0, elem)] = 0.5 * (matrix[IDX(size, 0, elem)] + matrix[IDX(size + 1, 1, elem)]);
    matrix[IDX(size + 1, size + 1, elem)] = 0.5 * (matrix[IDX(size, size + 1, elem)] + matrix[IDX(size + 1, size, elem)]);
}

__global__ void diffuse(int size, int idxOfElem, int b, float *state, float *state_prev, float diff, float dt, int num_of_indeces_block, int num_of_indeces_thread)
{
    int start_idx = num_of_indeces_block * blockIdx.x + num_of_indeces_thread * threadIdx.x;
    int end_idx = ((start_idx + num_of_indeces_thread) <= size) ? start_idx + num_of_indeces_thread : 0;
    if (end_idx == size)
        end_idx++;
    if (start_idx == 0)
        start_idx++;
    int row, col, k;
    float a = dt * diff * size * size;
    for (k = 0; k < 20; k++)
    {
        for (col = 1; col <= size; col++)
        {
            for (row = start_idx; row < end_idx; row++)
            {
                state[IDX(col, row, idxOfElem)] =
                    (state_prev[IDX(col, row, idxOfElem)] + a * (state[IDX(col - 1, row, idxOfElem)] +
                                                                 state[IDX(col + 1, row, idxOfElem)] +
                                                                 state[IDX(col, row - 1, idxOfElem)] +
                                                                 state[IDX(col, row + 1, idxOfElem)])) /
                    (1 + 4 * a);
            }
            __syncthreads();
        }
        __syncthreads();
        set_bnd(size, b, state, idxOfElem);
    }
}
__global__ void advect(int size, int idxOfEl_d, int b, float *state, float *state_prev, float *matrix_uv, float dt, int num_of_indeces_block, int num_of_indeces_thread)
{
    int start_idx = num_of_indeces_block * blockIdx.x + num_of_indeces_thread * threadIdx.x;
    int end_idx = ((start_idx + num_of_indeces_thread) <= size) ? start_idx + num_of_indeces_thread : 0;
    if (end_idx == size)
        end_idx++;
    if (start_idx == 0)
        start_idx++;
    int i0, j0, i1, j1;
    int row, col;
    float x, y, s0, t0, s1, t1, dt0;
    dt0 = dt * size;
    for (col = 1; col <= size; col++)
    {
        for (row = start_idx; row < end_idx; row++)
        {
            // calculating particle position based on u and v elems at time dt0
            x = col - dt0 * matrix_uv[IDX(col, row, ELEM_U)];
            y = row - dt0 * matrix_uv[IDX(col, row, ELEM_V)];
            if (x < 0.5) // Checking if particle X position isnt outside of grid
                x = 0.5; // Placing particle to grid border tile center
            if (x > size + 0.5)
                x = size + 0.5;
            i0 = (int)x; // Previous tile X position is same as centered particle X
            i1 = i0 + 1; // Current tile X
            if (y < 0.5)
                y = 0.5;
            if (y > size + 0.5)
                y = size + 0.5;
            j0 = (int)y;
            j1 = j0 + 1;
            s1 = x - i0;
            s0 = 1 - s1;
            t1 = y - j0;
            t0 = 1 - t1;
            state[IDX(col, row, idxOfEl_d)] =
                s0 * (t0 * state_prev[IDX(i0, j0, idxOfEl_d)] + t1 * state_prev[IDX(i0, j1, idxOfEl_d)]) +
                s1 * (t0 * state_prev[IDX(i1, j0, idxOfEl_d)] + t1 * state_prev[IDX(i1, j1, idxOfEl_d)]);
        }
        __syncthreads();
    }
    set_bnd(size, b, state, idxOfEl_d);
}

__global__ void project(int size, float *state, float *state_prev)
{
    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
        int row, col, k;
        float h;
        h = 1.0 / size;
        for (col = 1; col <= size; col++)
        {
            for (row = 1; row <= size; row++)
            {
                state_prev[IDX(col, row, ELEM_V)] =
                    -0.5 * h *
                    (state[IDX(col + 1, row, ELEM_U)] - state[IDX(col - 1, row, ELEM_U)] +
                     state[IDX(col, row + 1, ELEM_V)] - state[IDX(col, row - 1, ELEM_V)]);
                state_prev[IDX(col, row, ELEM_U)] = 0;
            }
        }
        set_bnd(size, 0, state_prev, ELEM_V);
        set_bnd(size, 0, state_prev, ELEM_U);

        for (k = 0; k < 20; k++)
        {
            for (col = 1; col <= size; col++)
            {
                for (row = 1; row <= size; row++)
                {
                    state_prev[IDX(col, row, ELEM_U)] =
                        (state_prev[IDX(col, row, ELEM_V)] +
                         state_prev[IDX(col - 1, row, ELEM_U)] + state_prev[IDX(col + 1, row, ELEM_U)] +
                         state_prev[IDX(col, row - 1, ELEM_U)] + state_prev[IDX(col, row + 1, ELEM_U)]) /
                        4;
                }
            }

            set_bnd(size, 0, state_prev, ELEM_U);
        }
        for (col = 1; col <= size; col++)
        {
            for (row = 1; row <= size; row++)
            {
                state[IDX(col, row, ELEM_U)] -=
                    0.5 * (state_prev[IDX(col + 1, row, ELEM_U)] - state_prev[IDX(col - 1, row, ELEM_U)]) / h;
                state[IDX(col, row, ELEM_V)] -=
                    0.5 * (state_prev[IDX(col, row + 1, ELEM_U)] - state_prev[IDX(col, row - 1, ELEM_U)]) / h;
            }
        }
        set_bnd(size, 1, state, ELEM_U);
        set_bnd(size, 2, state, ELEM_V);
    }
}

void swap(float *&a, float *&b)
{
    float *temp = a;
    a = b;
    b = temp;
}

int getItemFormArgInt(int argc, char *argv[], char a, int defaultV);
float getItemFormArgFloat(int argc, char *argv[], char a, float defaultV);
float Str2float10b(char str[]);

TCHAR szName[] = TEXT("sharedMemForFluidSim"); // Global

int main(int argc, char *argv[])
{
    int framesCreated = 1;
    double time_spend_avg_diff = 0;
    double time_spend_avg_vel = 0;
    int size = 0;
    size = getItemFormArgInt(argc, argv, 's', -1);
    if (size < 0)
    {
        printf("Size was never found in arguments,\n or there was an error during conversion\n");
        exit(-1);
    }
    int full_size = (size + 2) * (size + 2);
    const int comm_channels = 2;
    const int size_w_elem = full_size * ELEM_NUM;                   // Size of grid with spaces between tiles
    const int in_commands = size_w_elem;                            // Index where command from ui are located
    float diff_rate = getItemFormArgFloat(argc, argv, 'd', 0.001f); // diffusion rate
    float dt = getItemFormArgFloat(argc, argv, 'w', 0.001f);        // Time spacing between frames (snapshots)
    float *state;                                                   // Includes u, v, density
    float *state_prev;                                              // Includes u, v, density
    int blocks = getItemFormArgInt(argc, argv, 'b', 1);
    int threads = getItemFormArgInt(argc, argv, 't', 1);
    int mul_blocks_threads = (blocks * threads <= 0) ? 1 : blocks * threads;
    int num_of_indeces_thread = (int)ceil((float)(size) / (float)mul_blocks_threads);
    int num_of_indeces_block = num_of_indeces_thread * threads;
    num_of_indeces_block = (num_of_indeces_block == 0) ? 1 : num_of_indeces_block;
    const unsigned int sharedMemArrSize = size_w_elem + comm_channels;
    hipMallocManaged(&state, sharedMemArrSize * sizeof(float));
    hipMallocManaged(&state_prev, sharedMemArrSize * sizeof(float));

    state[IDX(size / 2, size / 2, ELEM_DENS)] = 1.0f;

    printf("{\"sharedMemSizeMB\":%.2f,", (float)(sharedMemArrSize * sizeof(float)) / (float)1000000);
    printf("\"sharedMemSizeB\":%d,", (int)(sharedMemArrSize * sizeof(float)));
    printf("\"n\":%d,\"blocks\":%d,\"threads\":%d,\"diffRate\":%f,\"dt\":%f,", size, blocks, threads, diff_rate, dt);
    printf("\"buffLen\":%u,", sharedMemArrSize);
    HANDLE hMapFile;
    LPCTSTR pBuf;

    /*Creates file mapping object in memory and saves the handle to hMapFile*/
    hMapFile = OpenFileMapping(
        FILE_MAP_ALL_ACCESS, // read/write access
        FALSE,               // do not inherit the name
        szName);             // name of mapping object

    if (hMapFile == NULL)
    {
        _tprintf(TEXT("Could not open file mapping object (%d).\n"),
                 GetLastError());
        return 1;
    }
    /*tries to connect mapping object to memory of this process*/
    pBuf = (LPTSTR)MapViewOfFile(hMapFile,            // handle to map object
                                 FILE_MAP_ALL_ACCESS, // read/write permission
                                 0,
                                 0,
                                 (int)(size_w_elem * sizeof(float)));
    /*Checks if connection attempt was successful*/
    if (pBuf == NULL)
    {
        _tprintf(TEXT("Could not map view of file (%d).\n"),
                 GetLastError());

        CloseHandle(hMapFile);

        return 1;
    }

    bool run = true;
    clock_t begin_of_mesurement = clock();
    clock_t end_of_mesurement = clock();
    while (run)
    {
        // getchar();
        while (state[in_commands] == NO_MESSAGE)
        {
            hipMemcpy(state, pBuf, (int)(sharedMemArrSize * sizeof(float)), hipMemcpyHostToDevice);
            if (state[in_commands] == STOP_ENGINE)
                run = false;
            if (state[in_commands] == NEXT_FRAME)
            {
                state[in_commands] = NO_MESSAGE;
                hipMemcpy((PVOID)pBuf, state, (int)(sharedMemArrSize * sizeof(float)), hipMemcpyDeviceToHost);
                break;
            }
            // printf("%f - %f\n", state[in_commands], STOP_ENGINE);
            Sleep(10);
        }
        // Velocity steps
        begin_of_mesurement = clock();
        swap(state, state_prev);
        diffuse<<<blocks, threads>>>(size, ELEM_U, 1, state, state_prev, diff_rate, dt, num_of_indeces_block, num_of_indeces_thread);
        diffuse<<<blocks, threads>>>(size, ELEM_V, 2, state, state_prev, diff_rate, dt, num_of_indeces_block, num_of_indeces_thread);
        hipDeviceSynchronize();
        project<<<blocks, threads>>>(size, state, state_prev);
        hipDeviceSynchronize();
        swap(state, state_prev);
        advect<<<blocks, threads>>>(size, ELEM_U, 1, state, state_prev, state_prev, diff_rate, num_of_indeces_block, num_of_indeces_thread);
        hipDeviceSynchronize();
        advect<<<blocks, threads>>>(size, ELEM_V, 2, state, state_prev, state_prev, diff_rate, num_of_indeces_block, num_of_indeces_thread);
        hipDeviceSynchronize();
        project<<<blocks, threads>>>(size, state, state_prev);
        hipDeviceSynchronize();
        end_of_mesurement = clock();
        time_spend_avg_vel += (double)(end_of_mesurement - begin_of_mesurement) * 1000000.0 / CLOCKS_PER_SEC;
        // Density steps
        begin_of_mesurement = clock();
        swap(state, state_prev);                                                                                                         // Swaping current state to previous
        diffuse<<<blocks, threads>>>(size, ELEM_DENS, 0, state, state_prev, diff_rate, dt, num_of_indeces_block, num_of_indeces_thread); // Calculate diffusion
        hipDeviceSynchronize();
        swap(state, state_prev);                                                                                                           // Swaping current state to previous
        advect<<<blocks, threads>>>(size, ELEM_DENS, 0, state, state_prev, state, diff_rate, num_of_indeces_block, num_of_indeces_thread); // Moving density
        hipDeviceSynchronize();
        end_of_mesurement = clock();

        time_spend_avg_diff += (double)(end_of_mesurement - begin_of_mesurement) * 1000000.0 / CLOCKS_PER_SEC;

        hipMemcpy((PVOID)pBuf, state, (int)(size_w_elem * sizeof(float)), hipMemcpyDeviceToHost);
        //  printf("%d - %s\n %s\n ---\n", hipGetLastError(), hipGetErrorName(hipGetLastError()), hipGetErrorString(hipGetLastError()));
        framesCreated++;
    }

    UnmapViewOfFile(pBuf);
    CloseHandle(hMapFile);

    // Free memory
    hipFree(state);
    hipFree(state_prev);

    time_spend_avg_diff /= framesCreated;
    int sec = time_spend_avg_diff / 1000000;
    time_spend_avg_diff = time_spend_avg_diff - 1000000 * sec;
    int ms = time_spend_avg_diff / 1000;
    time_spend_avg_diff = time_spend_avg_diff - 1000 * ms;
    printf("\"diffusion\":[\"%d\",\"%d\",\"%03d\"],", sec, ms, (int)time_spend_avg_diff);
    time_spend_avg_vel /= framesCreated;
    sec = time_spend_avg_vel / 1000000;
    time_spend_avg_vel = time_spend_avg_vel - 1000000 * sec;
    ms = time_spend_avg_vel / 1000;
    time_spend_avg_vel = time_spend_avg_vel - 1000 * ms;
    printf("\"velocity\":[\"%d\",\"%d\",\"%03d\"],", sec, ms, (int)time_spend_avg_vel);
    printf("\"frames\":%d}", framesCreated);
    return 0;
}

int getItemFormArgInt(int argc, char *argv[], char a = 's', int defaultV = -1)
{
    int i;
    int numberOf = 0;
    int found = false;
    for (i = 0; i < argc; i++)
    {
        if (strlen(argv[i]) > 3 && argv[i][0] == '-' && argv[i][1] == a && argv[i][2] == '=')
        {
            found = true;
            break;
        }
    }
    if (!found)
        return defaultV;
    char *token;
    const char s[2] = "=";
    // printf("%s\n", argv[i]);
    token = strtok(argv[i], s);
    while (token != NULL)
    {
        int converted = strtol(token, (char **)NULL, 10);
        if (converted > 0)
        {
            numberOf = converted;
            return numberOf;
        }
        token = strtok(NULL, s);
    }
    return (numberOf > 0) ? numberOf : defaultV;
}
float getItemFormArgFloat(int argc, char *argv[], char a = 's', float defaultV = 0.01)
{
    int i;
    float numberOf = 0;
    int found = false;
    for (i = 0; i < argc; i++)
    {
        if (strlen(argv[i]) > 3 && argv[i][0] == '-' && argv[i][1] == a && argv[i][2] == '=')
        {
            found = true;
            break;
        }
    }
    if (!found)
        return defaultV;
    char *token;
    const char s[2] = "=";
    // printf("%s\n", argv[i]);
    token = strtok(argv[i], s);
    while (token != NULL)
    {
        float converted = Str2float10b(token);
        if (converted > 0)
        {
            numberOf = converted;
            return numberOf;
        }
        token = strtok(NULL, s);
    }
    return (numberOf > 0) ? numberOf : defaultV;
}

float Str2float10b(char str[])
{
    int str_size = strlen(str);
    int floating_point_idx = 0;
    char *float_p;
    float_p = strchr(str, '.');
    floating_point_idx = (int)(float_p - str);

    char before_fp[50];
    char after_fp[50];

    int before_idx = 0;
    int after_idx = 0;
    for (int str_idx = 0; str_idx < str_size; str_idx++)
    {
        if (str_idx < floating_point_idx)
        {
            before_fp[before_idx] = str[str_idx];
            before_idx++;
            before_fp[before_idx] = '\0';
        }
        else if (str_idx > floating_point_idx)
        {
            after_fp[after_idx] = str[str_idx];
            after_idx++;
            after_fp[after_idx] = '\0';
        }
    }

    float result = (float)strtol(before_fp, (char **)NULL, 10);
    float decimal = (float)strtol(after_fp, (char **)NULL, 10) / pow(10, strlen(after_fp));
    result += (str[0] == '-') ? -decimal : decimal;
    return result;
}